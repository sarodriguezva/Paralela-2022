
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>

using namespace std;

__global__ void multiply_matrices(float *A, float *B, float *C, int size){
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;

    //TODO: Assign load
    if (x < size && y < size){
        for (int i = 0; i < size; i++){
            for (int j = 0; j < size; j++){
                for (int k = 0; k < size; k++){
                    C[i*size + j] += A[i*size + k] * B[k*size + j];
                }
            }
        }
    }
}

void print_matrix(float *matrix, int size, char name){
    cout << name << ":" << endl;
    for (int i = 0; i < size; i++){
        cout << "[ ";
        for (int j = 0; j < size; j++){
            cout << matrix[i*size + j] << " ";
        }
        cout << "]" << endl;
    }
    cout << endl;
}

int main(int argc, char *argv[]) {
    //Receives m, n, p. Matrices dimensions.
    int n = stoi(argv[1]);
    int THREADS = stoi(argv[2]);
    int num_blocks = stoi(argv[3]);
    int size = n*n*sizeof(float);

    srand(time(0));
    
    float *A = (float *) malloc(size);
    float *B = (float *) malloc(size);
    float *C = (float *) malloc(size);

    //Initialize matrices
    //A[i][j] = A[i*ncols + j]
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            A[i*n + j] = (float) rand()/RAND_MAX;
            B[i*n + j] = (float) rand()/RAND_MAX;
            C[i*n + j] = 0;
        }
    }

    print_matrix(A, n, 'A');
    print_matrix(B, n, 'B');

    //CUDA WORK
    float *d_A , *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, &A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, &B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, &C, size, hipMemcpyHostToDevice);

    multiply_matrices<<<num_blocks, THREADS>>>(d_A, d_B, d_C, n);

    hipMemcpy(&C, d_C, n, hipMemcpyDeviceToHost);

    print_matrix(C, n, 'C');
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    
    return 0;
}
