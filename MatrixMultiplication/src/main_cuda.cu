
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <iostream>
#include <math.h>

using namespace std;

__global__ void multiply_matrices(float *A, float *B, float *C, int n){
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;

    if (COL < n && ROW < n){
        float sum = 0;
        for (int k = 0; k < n; k++){
            sum += A[ROW*n + k] * B[k*n + COL];
        }
        C[ROW*n + COL] = sum;
    }
}

void print_matrix(float *matrix, int size, char name){
    cout << name << ":" << endl;
    for (int i = 0; i < size; i++){
        cout << "[ ";
        for (int j = 0; j < size; j++){
            cout << matrix[i*size + j] << " ";
        }
        cout << "]" << endl;
    }
    cout << endl;
}

int main(int argc, char *argv[]) {
    //Receives m, n, p. Matrices dimensions.
    int n = stoi(argv[1]);
    int THREADS = stoi(argv[2]);
    int size = n*n*sizeof(float);

    srand(time(0));
    
    float *A = (float *) malloc(size);
    float *B = (float *) malloc(size);
    float *C = (float *) malloc(size);

    //Initialize matrices
    //A[i][j] = A[i*ncols + j]
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            A[i*n + j] = (float) rand()/RAND_MAX;
            B[i*n + j] = (float) rand()/RAND_MAX;
            C[i*n + j] = 0;
        }
    }

    //CUDA WORK
    //n*n threads distributed in n/THREADS blocks, each block has THREADS*THREADS threads.
    float *d_A , *d_B, *d_C;
    dim3 threads_per_block(THREADS, THREADS);
    dim3 num_blocks(ceil((double) n / threads_per_block.x), ceil((double) n / threads_per_block.y));
    
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    multiply_matrices<<<num_blocks, threads_per_block>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    print_matrix(A, n, 'A');
    print_matrix(B, n, 'B');
    print_matrix(C, n, 'C');
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
    
    return 0;
}
